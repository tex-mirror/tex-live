#include "hip/hip_runtime.h"
#include "ReflectanceMapper.cuh"

#include "helper.cuh"
#include "utils.cuh"

#include <hip/hip_runtime.h>
#include <>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

__device__
inline float swap_bits(uint32_t const& x, uint32_t const& mask_1, unsigned int const& shft)
{
    return ((x & mask_1) << shft) | ((x & (~mask_1)) >> shft);
}

__device__ constexpr float recvbit = 2.32830643654e-10; // 1/2^32.
__device__ constexpr int REFL_NUM_SAMPLES = 1 << 15;

__device__
float van_der_corput_bitshift(uint32_t bits)
{
    bits = swap_bits(bits, 0x55555555, 1);
    bits = swap_bits(bits, 0x33333333, 2);
    bits = swap_bits(bits, 0x0F0F0F0F, 4);
    bits = swap_bits(bits, 0x00FF00FF, 8);
    bits = swap_bits(bits, 0x0000FFFF, 16);

    return static_cast<float>(bits) * recvbit;
}

__device__
glm::vec2 hamersely(uint32_t i, uint32_t N)
{
    return glm::vec2(static_cast<float>(i) / N, van_der_corput_bitshift(i));
}

__device__
glm::vec3 importance_sampl_GGX(glm::vec2 sample, glm::vec3 normal, float roughness)
{
    float a = roughness * roughness;

    float phi = TAU * sample.x;
    float cosTheta = sqrtf((1.0f - sample.y) / (1.f + (a * a - 1.f) * sample.y)); // GGX Sample, inverse sampling?
    // TODO: Understand the derivation behind this cosTheta. It has something to do with GGX distribution, but how?
    float sinTheta = sqrtf(1.0f - cosTheta * cosTheta);

    glm::vec3 vec = from_sphcoord(phi, cosTheta, sinTheta);
    glm::vec3 N1(cosTheta * __cosf(phi), cosTheta * __sinf(phi), -1 * sinTheta);
    glm::vec3 N2(-1 * __sinf(phi), __cosf(phi), 0);

    glm::mat3 normalBasis(N1, N2, normal);
    return normalBasis * vec;
}


#pragma region mapReflectance
__global__
void map_reflectance(hipTextureObject_t tObj,
    int width, int height, float roughness,
    float3* out, int outWidth, int outHeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    glm::vec3 result(0.0f);

    if (idx < outWidth && idx_y < outHeight)
    {
        int access_idx = to_idx(outWidth, idx, idx_y);

        float target_phi = TAU * ((idx + 0.5f) / outWidth);
        float target_theta = PI * ((idx_y + 0.5f) / outHeight);
        glm::vec3 N = from_sphcoord(target_phi, target_theta);

        float total_weight = 0.0f;
        for (int i = 0; i < REFL_NUM_SAMPLES; ++i)
        {
            glm::vec2 sample = hamersely(i, REFL_NUM_SAMPLES);
            glm::vec3 half_vec = importance_sampl_GGX(sample, N, roughness);

            // use the structure of rhombus to calculate lightvec
            glm::vec3 scaled_half = 2.0f * glm::dot(half_vec, N) * half_vec;
            glm::vec3 lightvec = glm::normalize(scaled_half - N);

            float ndotl = glm::dot(N, lightvec);
            glm::vec2 sphcoord = to_sphcoord(lightvec);
            float4 frag = tex2D<float4>(tObj,
                sphcoord.x * PI_RECR * width / 2,
                sphcoord.y * PI_RECR * height);

            glm::vec3 frag3(frag.x, frag.y, frag.z);

            if (ndotl > 0.0)
            {
                // epic games said it gives better results, otherwise weight can be set to 1.
#ifndef SET_WEIGHT_ONE
                float weight = ndotl;
#else
                float weight = 1.0f;
#endif
                result += (frag3 * weight);  // weighting by n@l, technically not required,
                total_weight += weight;
            }

        }
        if (total_weight > 0.0f)
        {
            result = result / total_weight;
            out[access_idx] = make_float3(result.x, result.y, result.z);
        }
        else
            out[access_idx] = make_float3(0, 0, 0);

    }
}

const size_t blkSz = 8;
void map_reflectance_ker(
    float4* in, float3* out, size_t width, size_t height, float roughness,
    size_t outWidth, size_t outHeight)
{
    float4* d_ptr;
    size_t pitch;
    cudaErrorCheck(hipMallocPitch(
        &d_ptr, &pitch, width * sizeof(float4), height));
    cudaErrorCheck(hipMemcpy2D(d_ptr, pitch, in,
        width * sizeof(float4), width * sizeof(float4),
        height, hipMemcpyHostToDevice));

    hipResourceDesc cRD;
    memset(&cRD, 0, sizeof(hipResourceDesc));
    cRD.resType = hipResourceTypePitch2D;
    cRD.res.pitch2D.devPtr = d_ptr;
    cRD.res.pitch2D.width = width;
    cRD.res.pitch2D.height = height;
    cRD.res.pitch2D.desc = hipCreateChannelDesc<float4>();
    cRD.res.pitch2D.pitchInBytes = pitch;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.sRGB = 0;
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t t_obj;
    cudaErrorCheck(hipCreateTextureObject(
        &t_obj, &cRD, &texDesc, nullptr));

    // out source
    float3* d_out = nullptr;
    cudaErrorCheck(hipMalloc(
        (void**)&d_out, static_cast<size_t>(sizeof(float3) * outWidth * outHeight)));
    dim3 blockSz((outWidth / blkSz) + 1, (outHeight / blkSz) + 1);
    dim3 kerSz(blkSz, blkSz);
    map_reflectance KERNEL_ARGS(blockSz, kerSz) (t_obj, width, height, roughness, d_out, outWidth, outHeight);

    cudaErrorCheck(hipMemcpy(
        out, d_out, sizeof(float3) * outWidth * outHeight, hipMemcpyDeviceToHost));

    cudaErrorCheck(hipDestroyTextureObject(t_obj));
    cudaErrorCheck(hipFree(d_ptr));
}


#pragma endregion

__device__
float norm_dist(float const& roughness, float3 const& half_vec)
{
    float alpha = roughness * roughness;
    float ndoth = half_vec.z; // assume N=(0,0,1)
    float denom_base = (ndoth * ndoth) * (alpha * alpha - 1) + 1;
    return (alpha * alpha) / (denom_base * denom_base * PI);
}

__device__
float G_component(float const& k, float const& ndotv)
{
    float denom = (ndotv * (1 - k)) + k;
    return 1 / denom;
}

__device__
float GFn(float const& roughness, float const& ndotl, float const& ndotv)
{
    float a = roughness * roughness;
    float k = a * a * 0.5;
    return G_component(k, ndotl) * G_component(k, ndotv);
}

__device__
float clamp(float const& x)
{
    return fminf(fmaxf(x, 0.0f), 1.0f);
}

// by symmetry, assume phi_v=0.
// porting of python code to CUDA

__device__ constexpr int LUT_INTEGRATE_SAMPLES = 8192;
__device__ constexpr float INTEGRATE_LUT_SCALE = 1.0f / LUT_INTEGRATE_SAMPLES;

__device__
float2 get_integrate_value(float const& roughness, float const& cos_theta)
{
    glm::vec2 value(0.0f);
    glm::vec3 upZ(0, 0, 1.0f);
    float num_samples = 0.0f;

    float cos_theta_v = clamp(cos_theta);
    float sin_theta_v = sqrtf(1 - cos_theta_v * cos_theta_v);
    glm::vec3 view_vec(sin_theta_v, 0, cos_theta_v);

    for (int i=0; i< LUT_INTEGRATE_SAMPLES; ++i)
    {
        glm::vec2 sample_coord = hamersely(i, LUT_INTEGRATE_SAMPLES);
        glm::vec3 half_vec = importance_sampl_GGX(sample_coord, upZ, roughness);

        glm::vec3 scaled_half = 2.0f * glm::dot(half_vec, view_vec) * half_vec;
        glm::vec3 lightvec = glm::normalize(scaled_half - view_vec);

        float ldotn = clamp(lightvec.z);
        float vdoth = clamp(glm::dot(half_vec, view_vec));
        float ndoth = clamp(half_vec.z);

        if (ldotn > 0.0f)
        {
            float base_val = (GFn(roughness, ldotn, cos_theta_v) * cos_theta_v * ldotn);
            float base_f = powf(1.0f - vdoth, 5.0f);
            
            value.x += base_val * (1 - base_f);
            value.y += base_val * base_f;
            num_samples += 1.0f;
        }
    }
    value = value * INTEGRATE_LUT_SCALE;

    return make_float2(value.x, value.y);
}

__global__
void generate_brdf_integrate(int width, int height, float2* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < width && idx_y < height)
    {
        int access_idx = to_idx(width, idx, idx_y);
        float cosv = (idx + 1.0f) / width;
        float roughness = (idx_y + 1.0f) / height;

        out[access_idx] = get_integrate_value(roughness, cosv);
    }
}

void generate_brdf_integrate_lut_ker(int width, int height, float2* out)
{
    float2* d_out;
    cudaErrorCheck(hipMalloc(
        (void**)&d_out, static_cast<size_t>(width * height * sizeof(float2))));


    dim3 blockSz((width / blkSz) + 1, (height / blkSz) + 1);
    dim3 kerSz(blkSz, blkSz);
    generate_brdf_integrate KERNEL_ARGS(blockSz, kerSz) (width, height, d_out);

    cudaErrorCheck(hipMemcpy(
        out, d_out, width * height * sizeof(float2), hipMemcpyDeviceToHost));
    cudaErrorCheck(hipFree(d_out));

}
