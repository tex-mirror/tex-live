#include "hip/hip_runtime.h"
﻿/**
* @file kernel.cu
* @author Supakorn "Jamie" Rassameemasmuang <jamievlin@outlook.com>
* CUDA Kernel for computing irradiance by solid angle integration

* Partially based on:
* https://cdn2.unrealengine.com/Resources/files/2013SiggraphPresentationsNotes-26915738.pdf
*/

#include "kernel.h"
#include "helper.cuh"
#include "utils.cuh"

#include "simpson.cuh"

#include <hip/hip_runtime.h>
#include <texture_indirect_functions.h>
#include <>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <functional>

class IntegrateSampler
{
public:
    __device__
    IntegrateSampler(
        hipTextureObject_t tObjin,
        glm::mat3 normalOrthBasis,
        size_t const& inWidth, size_t const& inHeight) :
        normalOrthBasis(normalOrthBasis), width(inWidth), height(inHeight),
        tObj(tObjin)

    {
    }

    __device__ ~IntegrateSampler() {}

    __device__
    glm::vec3 integrand(float const& sampled_phi, float const& sampled_theta)
    {
        // vec3 is the world space coordinate
        glm::vec2 sphcoord = to_sphcoord(angleToBasis(normalOrthBasis, sampled_phi, sampled_theta));
        float4 frag = tex2D<float4>(tObj,
            sphcoord.x * PI_RECR * 0.5*width,
            sphcoord.y * PI_RECR * height);

        return glm::vec3(frag.x, frag.y, frag.z);
    }

    __device__
    glm::vec3 inner(float const& sampled_theta)
    {
        return simpson(
          [this, &sampled_theta](float const& phi) {return this->integrand(phi,sampled_theta);  },
          0, TAU, acc)*0.5f*__sinf(2 * sampled_theta);
    }

    __device__
    glm::vec3 integrate()
    {
        return PI_RECR * simpson(
            [this](float const& theta) {return this->inner(theta); },
            0, HALFPI, acc);
    }

private:
    glm::mat3 normalOrthBasis;
    size_t width, height;
    hipTextureObject_t tObj;
};



__global__
void irradiate(hipTextureObject_t tObjin, float3* out, size_t width, size_t height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;


    if (idx < width && idx_y < height)
    {
        int access_idx = to_idx(width, idx, idx_y);

        float target_phi = TAU * ((idx + 0.5f) / width);
        float target_theta = PI * ((idx_y + 0.5f) / height);

        const glm::vec3 N = from_sphcoord(target_phi, target_theta);
        const glm::vec3 N1(
            __cosf(target_theta) * __cosf(target_phi),
            __cosf(target_theta) * __sinf(target_phi),
            -1*__sinf(target_theta));
        const glm::vec3 N2(-1 * __sinf(target_phi), __cosf(target_phi), 0);

        glm::mat3 normalBasisMat(N1,N2,N);

        IntegrateSampler integrator(tObjin, normalBasisMat, width, height);
        glm::vec3 out_val = integrator.integrate();
        out[access_idx] = make_float3(out_val.x, out_val.y, out_val.z);
    }
}

const size_t blkSz = 8;
void irradiate_ker(float4* in, float3* out, size_t width, size_t height)
{
    float4* d_ptr;
    size_t pitch;
    cudaErrorCheck(hipMallocPitch(
        &d_ptr, &pitch, width * sizeof(float4), height));
    cudaErrorCheck(hipMemcpy2D(d_ptr, pitch, in,
        width * sizeof(float4), width*sizeof(float4),
        height, hipMemcpyHostToDevice));

    hipResourceDesc cRD;
    memset(&cRD, 0, sizeof(hipResourceDesc));
    cRD.resType = hipResourceTypePitch2D;
    cRD.res.pitch2D.devPtr = d_ptr;
    cRD.res.pitch2D.width = width;
    cRD.res.pitch2D.height = height;
    cRD.res.pitch2D.desc = hipCreateChannelDesc<float4>();
    cRD.res.pitch2D.pitchInBytes = pitch;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.sRGB = 0;
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t t_obj;
    cudaErrorCheck(hipCreateTextureObject(
        &t_obj, &cRD, &texDesc, nullptr));

    // out source
    float3* d_out;
    cudaErrorCheck(hipMalloc(
        (void**)&d_out, static_cast<size_t>(sizeof(float3) * width * height)));
    dim3 blockSz((width / blkSz) + 1, (height / blkSz) + 1);
    dim3 kerSz(blkSz, blkSz);
    irradiate KERNEL_ARGS(blockSz, kerSz) (t_obj, d_out, width, height);

    cudaErrorCheck(hipMemcpy(
        out, d_out, sizeof(float3) * width * height, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipDestroyTextureObject(t_obj));
    hipFree(d_ptr);
}
